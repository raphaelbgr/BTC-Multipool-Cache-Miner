#include "hip/hip_runtime.h"
#include "cuda/engine.h"
#include "cuda/sha256d.cuh"

#include <hip/hip_runtime.h>

namespace cuda_engine {

__global__ void kernel_noop() {
  cuda_sha256d::hash256_once_stub();
}

void launchStub(const LaunchParams& params) {
  if (params.blocks == 0 || params.threads_per_block == 0) return;
  kernel_noop<<<params.blocks, params.threads_per_block>>>();
  hipDeviceSynchronize();
}

}  // namespace cuda_engine


