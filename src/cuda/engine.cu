#include "hip/hip_runtime.h"
#include "cuda/engine.h"
#include "cuda/sha256d.cuh"
#include "cuda/launch_plan.h"
#include <vector>

#include <hip/hip_runtime.h>

namespace cuda_engine {

__global__ void kernel_noop() {
  cuda_sha256d::hash256_once_stub();
}

__global__ void kernel_multi_noop() {
  // y-dimension indexes job, x-dimension covers blocks; no-op body for now
  cuda_sha256d::hash256_once_stub();
}

struct HitRecordDevice { unsigned long long work_id; unsigned int nonce; };

__global__ void kernel_write_hits(const unsigned long long* work_ids,
                                  unsigned int num_jobs,
                                  unsigned int nonce_base,
                                  HitRecordDevice* out_hits) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    out_hits[j].work_id = work_ids[j];
    out_hits[j].nonce = nonce_base + j;
  }
}

// Simple global buffers for a device-side ring
static __device__ unsigned int g_hit_write_idx = 0;
static HitRecordDevice* g_hit_buf = nullptr;
static unsigned int g_hit_cap = 0;
static DeviceJob* g_jobs = nullptr;
static unsigned int g_num_jobs = 0;

__global__ void kernel_init_hit_buf(HitRecordDevice* buf, unsigned int cap) {
  g_hit_buf = buf;
  g_hit_cap = cap;
  g_hit_write_idx = 0;
}

__global__ void kernel_push_hits(const unsigned long long* work_ids,
                                 unsigned int num_jobs,
                                 unsigned int nonce_base) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    unsigned int idx = atomicInc(&g_hit_write_idx, 0xFFFFFFFFu);
    unsigned int slot = (g_hit_cap == 0) ? 0u : (idx % g_hit_cap);
    if (g_hit_buf && g_hit_cap) {
      g_hit_buf[slot].work_id = work_ids[j];
      g_hit_buf[slot].nonce = nonce_base + j;
    }
  }
}

__global__ void kernel_mine_stub(unsigned int num_jobs, unsigned int nonce_base) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  // Assemble minimal header fields from g_jobs; when SHA implemented, compute hash and compare targets
  if (threadIdx.x == 0 && blockIdx.x == 0 && g_jobs && g_hit_buf && g_hit_cap) {
    // Assemble 80-byte big-endian header from DeviceJob (scaffold)
    unsigned char header[80];
    // version
    header[0] = (g_jobs[j].version >> 24) & 0xFF; header[1] = (g_jobs[j].version >> 16) & 0xFF;
    header[2] = (g_jobs[j].version >> 8) & 0xFF;  header[3] = (g_jobs[j].version) & 0xFF;
    // prevhash (convert LE words to BE bytes)
    #pragma unroll
    for (int w = 0; w < 8; ++w) {
      unsigned int v = g_jobs[j].prevhash_le[w];
      header[4 + w*4 + 0] = (v >> 24) & 0xFF;
      header[4 + w*4 + 1] = (v >> 16) & 0xFF;
      header[4 + w*4 + 2] = (v >> 8) & 0xFF;
      header[4 + w*4 + 3] = (v) & 0xFF;
    }
    // merkle root
    #pragma unroll
    for (int w = 0; w < 8; ++w) {
      unsigned int v = g_jobs[j].merkle_root_le[w];
      header[36 + w*4 + 0] = (v >> 24) & 0xFF;
      header[36 + w*4 + 1] = (v >> 16) & 0xFF;
      header[36 + w*4 + 2] = (v >> 8) & 0xFF;
      header[36 + w*4 + 3] = (v) & 0xFF;
    }
    // ntime clamp within [ntime_min, ntime_max]
    unsigned int ntime = g_jobs[j].ntime;
    if (g_jobs[j].ntime_min && ntime < g_jobs[j].ntime_min) ntime = g_jobs[j].ntime_min;
    if (g_jobs[j].ntime_max && ntime > g_jobs[j].ntime_max) ntime = g_jobs[j].ntime_max;
    header[68] = (ntime >> 24) & 0xFF; header[69] = (ntime >> 16) & 0xFF;
    header[70] = (ntime >> 8) & 0xFF;  header[71] = (ntime) & 0xFF;
    // nbits
    unsigned int nbits = g_jobs[j].nbits;
    header[72] = (nbits >> 24) & 0xFF; header[73] = (nbits >> 16) & 0xFF;
    header[74] = (nbits >> 8) & 0xFF;  header[75] = (nbits) & 0xFF;
    // nonce
    unsigned int nonce = nonce_base + j;
    header[76] = (nonce >> 24) & 0xFF; header[77] = (nonce >> 16) & 0xFF;
    header[78] = (nonce >> 8) & 0xFF;  header[79] = (nonce) & 0xFF;

    // Compute SHA256d and compare against share_target (LE u32[8])
    unsigned char digest[32];
    bool used_midstate = false;
    // If any word of midstate is non-zero, assume a valid midstate was provided
    #pragma unroll
    for (int i=0;i<8;++i) { if (g_jobs[j].midstate_le[i] != 0u) { used_midstate = true; break; } }
    if (used_midstate) {
      // Compute SHA256(header) using provided midstate for first 64 bytes
      unsigned int st[8];
      #pragma unroll
      for (int i=0;i<8;++i) st[i] = g_jobs[j].midstate_le[i];
      // Prepare second block words (16 words, BE) for bytes 64..79 + padding + len=640 bits
      unsigned int w0_15[16];
      // bytes 64..79 are header[64..79]
      #pragma unroll
      for (int i=0;i<4;++i) {
        int o = 64 + i*4;
        w0_15[i] = (unsigned int(header[o])<<24) | (unsigned int(header[o+1])<<16) |
                   (unsigned int(header[o+2])<<8) | (unsigned int(header[o+3]));
      }
      w0_15[4] = 0x80000000u; // 0x80 then zeros
      #pragma unroll
      for (int i=5;i<15;++i) w0_15[i] = 0u;
      w0_15[15] = 640u;
      cuda_sha256d::sha256_compress(st, w0_15);
      // First digest (big-endian) now in st
      unsigned char t1[32];
      #pragma unroll
      for (int i=0;i<8;++i) {
        t1[i*4+0] = (unsigned char)((st[i] >> 24) & 0xFF);
        t1[i*4+1] = (unsigned char)((st[i] >> 16) & 0xFF);
        t1[i*4+2] = (unsigned char)((st[i] >> 8) & 0xFF);
        t1[i*4+3] = (unsigned char)((st[i]) & 0xFF);
      }
      // Second SHA over 32-byte digest
      unsigned int st2[8];
      #pragma unroll
      for (int i=0;i<8;++i) st2[i] = cuda_sha256d::kSha256IV[i]; // access constant via namespace if visible
      unsigned int w2[16];
      #pragma unroll
      for (int i=0;i<8;++i) {
        int o = i*4;
        w2[i] = (unsigned int(t1[o])<<24) | (unsigned int(t1[o+1])<<16) |
                (unsigned int(t1[o+2])<<8) | (unsigned int(t1[o+3]));
      }
      w2[8] = 0x80000000u; for (int i=9;i<15;++i) w2[i]=0u; w2[15] = 256u;
      cuda_sha256d::sha256_compress(st2, w2);
      // Output digest
      #pragma unroll
      for (int i=0;i<8;++i) {
        digest[i*4+0] = (unsigned char)((st2[i] >> 24) & 0xFF);
        digest[i*4+1] = (unsigned char)((st2[i] >> 16) & 0xFF);
        digest[i*4+2] = (unsigned char)((st2[i] >> 8) & 0xFF);
        digest[i*4+3] = (unsigned char)((st2[i]) & 0xFF);
      }
    } else {
      cuda_sha256d::sha256d_80_be(header, digest);
    }
    // Compare big-endian digest to precomputed big-endian share target
    bool leq = true;
    #pragma unroll
    for (int i=0;i<32;++i) {
      unsigned char t = g_jobs[j].share_target_be[i];
      if (digest[i] < t) { leq = true; break; }
      if (digest[i] > t) { leq = false; break; }
    }
    if (leq) {
      unsigned int idx = atomicInc(&g_hit_write_idx, 0xFFFFFFFFu);
      unsigned int slot = (g_hit_cap == 0) ? 0u : (idx % g_hit_cap);
      g_hit_buf[slot].work_id = g_jobs[j].work_id;
      g_hit_buf[slot].nonce = nonce;
    }
  }
}

__global__ void kernel_mine_batch(unsigned int num_jobs, unsigned int nonce_base, unsigned int nonces_per_thread) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  unsigned int lane = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int start_nonce = nonce_base + j + lane * nonces_per_thread;
  // Iterate micro-batch per thread
  for (unsigned int k = 0; k < nonces_per_thread; ++k) {
    unsigned int nonce = start_nonce + k;
    if (threadIdx.x == 0 && blockIdx.x == 0) {
      // Reuse single-thread path for header assembly and hashing
      // Minimal duplication: call same body by inlining the key section
      unsigned char header[80];
      header[0] = (g_jobs[j].version >> 24) & 0xFF; header[1] = (g_jobs[j].version >> 16) & 0xFF;
      header[2] = (g_jobs[j].version >> 8) & 0xFF;  header[3] = (g_jobs[j].version) & 0xFF;
      #pragma unroll
      for (int w = 0; w < 8; ++w) {
        unsigned int v = g_jobs[j].prevhash_le[w];
        header[4 + w*4 + 0] = (v >> 24) & 0xFF;
        header[4 + w*4 + 1] = (v >> 16) & 0xFF;
        header[4 + w*4 + 2] = (v >> 8) & 0xFF;
        header[4 + w*4 + 3] = (v) & 0xFF;
      }
      #pragma unroll
      for (int w = 0; w < 8; ++w) {
        unsigned int v = g_jobs[j].merkle_root_le[w];
        header[36 + w*4 + 0] = (v >> 24) & 0xFF;
        header[36 + w*4 + 1] = (v >> 16) & 0xFF;
        header[36 + w*4 + 2] = (v >> 8) & 0xFF;
        header[36 + w*4 + 3] = (v) & 0xFF;
      }
      unsigned int ntime = g_jobs[j].ntime;
      if (g_jobs[j].ntime_min && ntime < g_jobs[j].ntime_min) ntime = g_jobs[j].ntime_min;
      if (g_jobs[j].ntime_max && ntime > g_jobs[j].ntime_max) ntime = g_jobs[j].ntime_max;
      header[68] = (ntime >> 24) & 0xFF; header[69] = (ntime >> 16) & 0xFF;
      header[70] = (ntime >> 8) & 0xFF;  header[71] = (ntime) & 0xFF;
      unsigned int nbits = g_jobs[j].nbits;
      header[72] = (nbits >> 24) & 0xFF; header[73] = (nbits >> 16) & 0xFF;
      header[74] = (nbits >> 8) & 0xFF;  header[75] = (nbits) & 0xFF;
      header[76] = (nonce >> 24) & 0xFF; header[77] = (nonce >> 16) & 0xFF;
      header[78] = (nonce >> 8) & 0xFF;  header[79] = (nonce) & 0xFF;
      unsigned char digest[32];
      cuda_sha256d::sha256d_80_be(header, digest);
      bool leq = true;
      #pragma unroll
      for (int i=0;i<32;++i) {
        unsigned char t = g_jobs[j].share_target_be[i];
        if (digest[i] < t) { leq = true; break; }
        if (digest[i] > t) { leq = false; break; }
      }
      if (leq) {
        unsigned int idx = atomicInc(&g_hit_write_idx, 0xFFFFFFFFu);
        unsigned int slot = (g_hit_cap == 0) ? 0u : (idx % g_hit_cap);
        g_hit_buf[slot].work_id = g_jobs[j].work_id;
        g_hit_buf[slot].nonce = nonce;
      }
    }
  }
}
__global__ void kernel_hash_one(uint32_t job_index, uint32_t nonce, unsigned char* out32) {
  if (!g_jobs) return;
  unsigned int j = job_index;
  unsigned char header[80];
  // version
  header[0] = (g_jobs[j].version >> 24) & 0xFF; header[1] = (g_jobs[j].version >> 16) & 0xFF;
  header[2] = (g_jobs[j].version >> 8) & 0xFF;  header[3] = (g_jobs[j].version) & 0xFF;
  // prevhash
  #pragma unroll
  for (int w = 0; w < 8; ++w) {
    unsigned int v = g_jobs[j].prevhash_le[w];
    header[4 + w*4 + 0] = (v >> 24) & 0xFF;
    header[4 + w*4 + 1] = (v >> 16) & 0xFF;
    header[4 + w*4 + 2] = (v >> 8) & 0xFF;
    header[4 + w*4 + 3] = (v) & 0xFF;
  }
  // merkle
  #pragma unroll
  for (int w = 0; w < 8; ++w) {
    unsigned int v = g_jobs[j].merkle_root_le[w];
    header[36 + w*4 + 0] = (v >> 24) & 0xFF;
    header[36 + w*4 + 1] = (v >> 16) & 0xFF;
    header[36 + w*4 + 2] = (v >> 8) & 0xFF;
    header[36 + w*4 + 3] = (v) & 0xFF;
  }
  // ntime
  unsigned int ntime = g_jobs[j].ntime;
  if (g_jobs[j].ntime_min && ntime < g_jobs[j].ntime_min) ntime = g_jobs[j].ntime_min;
  if (g_jobs[j].ntime_max && ntime > g_jobs[j].ntime_max) ntime = g_jobs[j].ntime_max;
  header[68] = (ntime >> 24) & 0xFF; header[69] = (ntime >> 16) & 0xFF;
  header[70] = (ntime >> 8) & 0xFF;  header[71] = (ntime) & 0xFF;
  // nbits
  unsigned int nbits = g_jobs[j].nbits;
  header[72] = (nbits >> 24) & 0xFF; header[73] = (nbits >> 16) & 0xFF;
  header[74] = (nbits >> 8) & 0xFF;  header[75] = (nbits) & 0xFF;
  // nonce
  header[76] = (nonce >> 24) & 0xFF; header[77] = (nonce >> 16) & 0xFF;
  header[78] = (nonce >> 8) & 0xFF;  header[79] = (nonce) & 0xFF;
  unsigned char digest[32];
  cuda_sha256d::sha256d_80_be(header, digest);
  // write digest to out32
  for (int i=0;i<32;++i) out32[i] = digest[i];
}

bool cuda_engine::computeDeviceHashForJob(uint32_t job_index, uint32_t nonce, unsigned char out32_host[32]) {
#ifndef __HIPCC__
  (void)job_index; (void)nonce; (void)out32_host; return true;
#else
  if (!g_jobs) return false;
  unsigned char* d_out = nullptr;
  hipMalloc(&d_out, 32);
  kernel_hash_one<<<1,1>>>(job_index, nonce, d_out);
  hipDeviceSynchronize();
  hipMemcpy(out32_host, d_out, 32, hipMemcpyDeviceToHost);
  hipFree(d_out);
  return true;
#endif
}

void launchStub(const LaunchParams& params) {
  if (params.blocks == 0 || params.threads_per_block == 0) return;
  kernel_noop<<<params.blocks, params.threads_per_block>>>();
  hipDeviceSynchronize();
}

bool cuda_engine::launchMultiJobStub(uint32_t num_jobs, uint64_t desired_threads_per_job) {
#ifndef __HIPCC__
  (void)num_jobs; (void)desired_threads_per_job;
  return false;
#else
  auto plan = computeLaunchPlan(num_jobs, desired_threads_per_job);
  if (plan.num_jobs == 0 || plan.blocks_per_job == 0 || plan.threads_per_block == 0) return false;
  dim3 grid(plan.blocks_per_job, plan.num_jobs, 1);
  dim3 block(plan.threads_per_block, 1, 1);
  kernel_multi_noop<<<grid, block>>>();
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::launchMineStub(uint32_t num_jobs, uint32_t nonce_base) {
#ifndef __HIPCC__
  (void)num_jobs; (void)nonce_base; return true;
#else
  if (g_num_jobs == 0 || num_jobs == 0) return false;
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_mine_stub<<<grid, block>>>(num_jobs, nonce_base);
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::launchMineWithPlan(uint32_t num_jobs,
                                     uint32_t blocks_per_job,
                                     uint32_t threads_per_block,
                                     uint32_t nonce_base) {
#ifndef __HIPCC__
  (void)num_jobs; (void)blocks_per_job; (void)threads_per_block; (void)nonce_base; return true;
#else
  if (g_num_jobs == 0 || num_jobs == 0) return false;
  if (blocks_per_job == 0 || threads_per_block == 0) return false;
  dim3 grid(blocks_per_job, num_jobs, 1);
  dim3 block(threads_per_block, 1, 1);
  kernel_mine_stub<<<grid, block>>>(num_jobs, nonce_base);
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::launchMineWithPlanBatch(uint32_t num_jobs,
                                          uint32_t blocks_per_job,
                                          uint32_t threads_per_block,
                                          uint32_t nonce_base,
                                          uint32_t nonces_per_thread) {
#ifndef __HIPCC__
  (void)num_jobs; (void)blocks_per_job; (void)threads_per_block; (void)nonce_base; (void)nonces_per_thread; return true;
#else
  if (g_num_jobs == 0 || num_jobs == 0) return false;
  if (blocks_per_job == 0 || threads_per_block == 0) return false;
  if (nonces_per_thread == 0) nonces_per_thread = 1;
  dim3 grid(blocks_per_job, num_jobs, 1);
  dim3 block(threads_per_block, 1, 1);
  kernel_mine_batch<<<grid, block>>>(num_jobs, nonce_base, nonces_per_thread);
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::uploadDeviceJobs(const DeviceJob* jobs_host, uint32_t num_jobs) {
#ifndef __HIPCC__
  (void)jobs_host; (void)num_jobs; return true;
#else
  if (num_jobs == 0) return false;
  if (g_jobs) hipFree(g_jobs);
  hipMalloc(&g_jobs, sizeof(DeviceJob) * num_jobs);
  hipMemcpy(g_jobs, jobs_host, sizeof(DeviceJob) * num_jobs, hipMemcpyHostToDevice);
  g_num_jobs = num_jobs;
  return true;
#endif
}

bool cuda_engine::launchWriteHitsDemo(const uint64_t* work_ids_host,
                                      uint32_t num_jobs,
                                      uint32_t nonce_base,
                                      HitRecord* out_hits_host) {
#ifndef __HIPCC__
  // Fallback: fill on host for non-CUDA builds
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = work_ids_host[j];
    out_hits_host[j].nonce = nonce_base + j;
  }
  return true;
#else
  if (num_jobs == 0) return false;
  unsigned long long* d_work_ids = nullptr;
  HitRecordDevice* d_hits = nullptr;
  hipMalloc(&d_work_ids, sizeof(unsigned long long) * num_jobs);
  hipMalloc(&d_hits, sizeof(HitRecordDevice) * num_jobs);
  hipMemcpy(d_work_ids, work_ids_host, sizeof(unsigned long long) * num_jobs, hipMemcpyHostToDevice);
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_write_hits<<<grid, block>>>(d_work_ids, num_jobs, nonce_base, d_hits);
  hipDeviceSynchronize();
  // Copy back
  std::vector<HitRecordDevice> tmp(num_jobs);
  hipMemcpy(tmp.data(), d_hits, sizeof(HitRecordDevice) * num_jobs, hipMemcpyDeviceToHost);
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = static_cast<uint64_t>(tmp[j].work_id);
    out_hits_host[j].nonce = static_cast<uint32_t>(tmp[j].nonce);
  }
  hipFree(d_hits);
  hipFree(d_work_ids);
  return true;
#endif
}

static HitRecordDevice* s_device_hit_buf = nullptr;
static unsigned int s_device_hit_cap = 0;
static unsigned int s_device_drain_offset = 0; // host-side read position (monotonic)

bool cuda_engine::initDeviceHitBuffer(uint32_t capacity) {
#ifndef __HIPCC__
  (void)capacity; return true;
#else
  if (capacity == 0) capacity = 1;
  if (s_device_hit_buf) hipFree(s_device_hit_buf);
  s_device_hit_cap = capacity;
  hipMalloc(&s_device_hit_buf, sizeof(HitRecordDevice) * capacity);
  kernel_init_hit_buf<<<1,1>>>(s_device_hit_buf, capacity);
  hipDeviceSynchronize();
  s_device_drain_offset = 0;
  return true;
#endif
}

void cuda_engine::freeDeviceHitBuffer() {
#ifdef __HIPCC__
  if (s_device_hit_buf) hipFree(s_device_hit_buf);
  s_device_hit_buf = nullptr;
  s_device_hit_cap = 0;
#endif
}

bool cuda_engine::launchPushHitsToDeviceRing(const uint64_t* work_ids_host,
                                             uint32_t num_jobs,
                                             uint32_t nonce_base) {
#ifndef __HIPCC__
  (void)work_ids_host; (void)num_jobs; (void)nonce_base; return true;
#else
  if (!s_device_hit_buf || s_device_hit_cap == 0) return false;
  if (num_jobs == 0) return true;
  unsigned long long* d_work_ids = nullptr;
  hipMalloc(&d_work_ids, sizeof(unsigned long long) * num_jobs);
  hipMemcpy(d_work_ids, work_ids_host, sizeof(unsigned long long) * num_jobs, hipMemcpyHostToDevice);
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_push_hits<<<grid, block>>>(d_work_ids, num_jobs, nonce_base);
  hipDeviceSynchronize();
  hipFree(d_work_ids);
  return true;
#endif
}

bool cuda_engine::drainDeviceHits(HitRecord* out_hits_host,
                                  uint32_t max_out,
                                  uint32_t* out_count) {
#ifndef __HIPCC__
  *out_count = 0; return true;
#else
  if (!s_device_hit_buf || s_device_hit_cap == 0) { *out_count = 0; return false; }
  // Read current write idx from device symbol
  unsigned int write_idx = 0;
  hipMemcpyFromSymbol(&write_idx, HIP_SYMBOL(g_hit_write_idx), sizeof(unsigned int));
  // Determine how many entries were written since last drain (modulo 2^32)
  unsigned int available = (write_idx >= s_device_drain_offset)
                             ? (write_idx - s_device_drain_offset)
                             : (0xFFFFFFFFu - s_device_drain_offset + 1u + write_idx);
  if (available == 0) { *out_count = 0; return true; }
  unsigned int to_copy = (available > max_out) ? max_out : available;
  // Copy from ring with wrap handling
  unsigned int start = s_device_drain_offset % s_device_hit_cap;
  unsigned int first = (to_copy < (s_device_hit_cap - start)) ? to_copy : (s_device_hit_cap - start);
  std::vector<HitRecordDevice> tmp(to_copy);
  if (first > 0) {
    hipMemcpy(tmp.data(), s_device_hit_buf + start, sizeof(HitRecordDevice) * first, hipMemcpyDeviceToHost);
  }
  if (first < to_copy) {
    hipMemcpy(tmp.data() + first, s_device_hit_buf, sizeof(HitRecordDevice) * (to_copy - first), hipMemcpyDeviceToHost);
  }
  for (unsigned int i = 0; i < to_copy; ++i) {
    out_hits_host[i].work_id = static_cast<uint64_t>(tmp[i].work_id);
    out_hits_host[i].nonce = static_cast<uint32_t>(tmp[i].nonce);
  }
  *out_count = to_copy;
  s_device_drain_offset += to_copy;
  return true;
#endif
}

}  // namespace cuda_engine


