#include "hip/hip_runtime.h"
#include "cuda/engine.h"
#include "cuda/sha256d.cuh"
#include "cuda/launch_plan.h"

#include <hip/hip_runtime.h>

namespace cuda_engine {

__global__ void kernel_noop() {
  cuda_sha256d::hash256_once_stub();
}

__global__ void kernel_multi_noop() {
  // y-dimension indexes job, x-dimension covers blocks; no-op body for now
  cuda_sha256d::hash256_once_stub();
}

struct HitRecordDevice { unsigned long long work_id; unsigned int nonce; };

__global__ void kernel_write_hits(const unsigned long long* work_ids,
                                  unsigned int num_jobs,
                                  unsigned int nonce_base,
                                  HitRecordDevice* out_hits) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    out_hits[j].work_id = work_ids[j];
    out_hits[j].nonce = nonce_base + j;
  }
}

void launchStub(const LaunchParams& params) {
  if (params.blocks == 0 || params.threads_per_block == 0) return;
  kernel_noop<<<params.blocks, params.threads_per_block>>>();
  hipDeviceSynchronize();
}

bool cuda_engine::launchMultiJobStub(uint32_t num_jobs, uint64_t desired_threads_per_job) {
#ifndef __HIPCC__
  (void)num_jobs; (void)desired_threads_per_job;
  return false;
#else
  auto plan = computeLaunchPlan(num_jobs, desired_threads_per_job);
  if (plan.num_jobs == 0 || plan.blocks_per_job == 0 || plan.threads_per_block == 0) return false;
  dim3 grid(plan.blocks_per_job, plan.num_jobs, 1);
  dim3 block(plan.threads_per_block, 1, 1);
  kernel_multi_noop<<<grid, block>>>();
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::launchWriteHitsDemo(const uint64_t* work_ids_host,
                                      uint32_t num_jobs,
                                      uint32_t nonce_base,
                                      HitRecord* out_hits_host) {
#ifndef __HIPCC__
  // Fallback: fill on host for non-CUDA builds
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = work_ids_host[j];
    out_hits_host[j].nonce = nonce_base + j;
  }
  return true;
#else
  if (num_jobs == 0) return false;
  unsigned long long* d_work_ids = nullptr;
  HitRecordDevice* d_hits = nullptr;
  hipMalloc(&d_work_ids, sizeof(unsigned long long) * num_jobs);
  hipMalloc(&d_hits, sizeof(HitRecordDevice) * num_jobs);
  hipMemcpy(d_work_ids, work_ids_host, sizeof(unsigned long long) * num_jobs, hipMemcpyHostToDevice);
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_write_hits<<<grid, block>>>(d_work_ids, num_jobs, nonce_base, d_hits);
  hipDeviceSynchronize();
  // Copy back
  std::vector<HitRecordDevice> tmp(num_jobs);
  hipMemcpy(tmp.data(), d_hits, sizeof(HitRecordDevice) * num_jobs, hipMemcpyDeviceToHost);
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = static_cast<uint64_t>(tmp[j].work_id);
    out_hits_host[j].nonce = static_cast<uint32_t>(tmp[j].nonce);
  }
  hipFree(d_hits);
  hipFree(d_work_ids);
  return true;
#endif
}

}  // namespace cuda_engine


