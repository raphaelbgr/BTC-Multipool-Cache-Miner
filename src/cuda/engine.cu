#include "hip/hip_runtime.h"
#include "cuda/engine.h"
#include "cuda/sha256d.cuh"
#include "cuda/launch_plan.h"
#include <vector>

#include <hip/hip_runtime.h>

namespace cuda_engine {

__global__ void kernel_noop() {
  cuda_sha256d::hash256_once_stub();
}

__global__ void kernel_multi_noop() {
  // y-dimension indexes job, x-dimension covers blocks; no-op body for now
  cuda_sha256d::hash256_once_stub();
}

struct HitRecordDevice { unsigned long long work_id; unsigned int nonce; };

__global__ void kernel_write_hits(const unsigned long long* work_ids,
                                  unsigned int num_jobs,
                                  unsigned int nonce_base,
                                  HitRecordDevice* out_hits) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    out_hits[j].work_id = work_ids[j];
    out_hits[j].nonce = nonce_base + j;
  }
}

// Simple global buffers for a device-side ring
static __device__ unsigned int g_hit_write_idx = 0;
static HitRecordDevice* g_hit_buf = nullptr;
static unsigned int g_hit_cap = 0;

__global__ void kernel_init_hit_buf(HitRecordDevice* buf, unsigned int cap) {
  g_hit_buf = buf;
  g_hit_cap = cap;
  g_hit_write_idx = 0;
}

__global__ void kernel_push_hits(const unsigned long long* work_ids,
                                 unsigned int num_jobs,
                                 unsigned int nonce_base) {
  unsigned int j = blockIdx.y;
  if (j >= num_jobs) return;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    unsigned int idx = atomicInc(&g_hit_write_idx, 0xFFFFFFFFu);
    unsigned int slot = (g_hit_cap == 0) ? 0u : (idx % g_hit_cap);
    if (g_hit_buf && g_hit_cap) {
      g_hit_buf[slot].work_id = work_ids[j];
      g_hit_buf[slot].nonce = nonce_base + j;
    }
  }
}

void launchStub(const LaunchParams& params) {
  if (params.blocks == 0 || params.threads_per_block == 0) return;
  kernel_noop<<<params.blocks, params.threads_per_block>>>();
  hipDeviceSynchronize();
}

bool cuda_engine::launchMultiJobStub(uint32_t num_jobs, uint64_t desired_threads_per_job) {
#ifndef __HIPCC__
  (void)num_jobs; (void)desired_threads_per_job;
  return false;
#else
  auto plan = computeLaunchPlan(num_jobs, desired_threads_per_job);
  if (plan.num_jobs == 0 || plan.blocks_per_job == 0 || plan.threads_per_block == 0) return false;
  dim3 grid(plan.blocks_per_job, plan.num_jobs, 1);
  dim3 block(plan.threads_per_block, 1, 1);
  kernel_multi_noop<<<grid, block>>>();
  hipDeviceSynchronize();
  return true;
#endif
}

bool cuda_engine::launchWriteHitsDemo(const uint64_t* work_ids_host,
                                      uint32_t num_jobs,
                                      uint32_t nonce_base,
                                      HitRecord* out_hits_host) {
#ifndef __HIPCC__
  // Fallback: fill on host for non-CUDA builds
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = work_ids_host[j];
    out_hits_host[j].nonce = nonce_base + j;
  }
  return true;
#else
  if (num_jobs == 0) return false;
  unsigned long long* d_work_ids = nullptr;
  HitRecordDevice* d_hits = nullptr;
  hipMalloc(&d_work_ids, sizeof(unsigned long long) * num_jobs);
  hipMalloc(&d_hits, sizeof(HitRecordDevice) * num_jobs);
  hipMemcpy(d_work_ids, work_ids_host, sizeof(unsigned long long) * num_jobs, hipMemcpyHostToDevice);
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_write_hits<<<grid, block>>>(d_work_ids, num_jobs, nonce_base, d_hits);
  hipDeviceSynchronize();
  // Copy back
  std::vector<HitRecordDevice> tmp(num_jobs);
  hipMemcpy(tmp.data(), d_hits, sizeof(HitRecordDevice) * num_jobs, hipMemcpyDeviceToHost);
  for (uint32_t j = 0; j < num_jobs; ++j) {
    out_hits_host[j].work_id = static_cast<uint64_t>(tmp[j].work_id);
    out_hits_host[j].nonce = static_cast<uint32_t>(tmp[j].nonce);
  }
  hipFree(d_hits);
  hipFree(d_work_ids);
  return true;
#endif
}

static HitRecordDevice* s_device_hit_buf = nullptr;
static unsigned int s_device_hit_cap = 0;
static unsigned int s_device_drain_offset = 0; // host-side read position (monotonic)

bool cuda_engine::initDeviceHitBuffer(uint32_t capacity) {
#ifndef __HIPCC__
  (void)capacity; return true;
#else
  if (capacity == 0) capacity = 1;
  if (s_device_hit_buf) hipFree(s_device_hit_buf);
  s_device_hit_cap = capacity;
  hipMalloc(&s_device_hit_buf, sizeof(HitRecordDevice) * capacity);
  kernel_init_hit_buf<<<1,1>>>(s_device_hit_buf, capacity);
  hipDeviceSynchronize();
  s_device_drain_offset = 0;
  return true;
#endif
}

void cuda_engine::freeDeviceHitBuffer() {
#ifdef __HIPCC__
  if (s_device_hit_buf) hipFree(s_device_hit_buf);
  s_device_hit_buf = nullptr;
  s_device_hit_cap = 0;
#endif
}

bool cuda_engine::launchPushHitsToDeviceRing(const uint64_t* work_ids_host,
                                             uint32_t num_jobs,
                                             uint32_t nonce_base) {
#ifndef __HIPCC__
  (void)work_ids_host; (void)num_jobs; (void)nonce_base; return true;
#else
  if (!s_device_hit_buf || s_device_hit_cap == 0) return false;
  if (num_jobs == 0) return true;
  unsigned long long* d_work_ids = nullptr;
  hipMalloc(&d_work_ids, sizeof(unsigned long long) * num_jobs);
  hipMemcpy(d_work_ids, work_ids_host, sizeof(unsigned long long) * num_jobs, hipMemcpyHostToDevice);
  dim3 grid(1, num_jobs, 1);
  dim3 block(64, 1, 1);
  kernel_push_hits<<<grid, block>>>(d_work_ids, num_jobs, nonce_base);
  hipDeviceSynchronize();
  hipFree(d_work_ids);
  return true;
#endif
}

bool cuda_engine::drainDeviceHits(HitRecord* out_hits_host,
                                  uint32_t max_out,
                                  uint32_t* out_count) {
#ifndef __HIPCC__
  *out_count = 0; return true;
#else
  if (!s_device_hit_buf || s_device_hit_cap == 0) { *out_count = 0; return false; }
  // Read current write idx from device symbol
  unsigned int write_idx = 0;
  hipMemcpyFromSymbol(&write_idx, HIP_SYMBOL(g_hit_write_idx), sizeof(unsigned int));
  // Determine how many entries were written since last drain (modulo 2^32)
  unsigned int available = (write_idx >= s_device_drain_offset)
                             ? (write_idx - s_device_drain_offset)
                             : (0xFFFFFFFFu - s_device_drain_offset + 1u + write_idx);
  if (available == 0) { *out_count = 0; return true; }
  unsigned int to_copy = (available > max_out) ? max_out : available;
  // Copy from ring with wrap handling
  unsigned int start = s_device_drain_offset % s_device_hit_cap;
  unsigned int first = (to_copy < (s_device_hit_cap - start)) ? to_copy : (s_device_hit_cap - start);
  std::vector<HitRecordDevice> tmp(to_copy);
  if (first > 0) {
    hipMemcpy(tmp.data(), s_device_hit_buf + start, sizeof(HitRecordDevice) * first, hipMemcpyDeviceToHost);
  }
  if (first < to_copy) {
    hipMemcpy(tmp.data() + first, s_device_hit_buf, sizeof(HitRecordDevice) * (to_copy - first), hipMemcpyDeviceToHost);
  }
  for (unsigned int i = 0; i < to_copy; ++i) {
    out_hits_host[i].work_id = static_cast<uint64_t>(tmp[i].work_id);
    out_hits_host[i].nonce = static_cast<uint32_t>(tmp[i].nonce);
  }
  *out_count = to_copy;
  s_device_drain_offset += to_copy;
  return true;
#endif
}

}  // namespace cuda_engine


